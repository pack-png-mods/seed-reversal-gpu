// IDE indexing
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __shared__
#define __constant__
#define __global__
#define __CUDACC__
#include <device_functions.h>
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_cmath.h>
#endif




#include <hip/hip_runtime.h>
#include <stdint.h>
#include <memory.h>
#include <stdio.h>
#include <time.h>


#define signed_seed_t int64_t
#define uint uint32_t
#define ulong uint64_t
// let's be EVIL (and make sure all includes come before this)
#define int int32_t

#undef JRAND_DOUBLE

#define RANDOM_MULTIPLIER_LONG 0x5DEECE66DULL

#ifdef JRAND_DOUBLE
#define Random double
#define RANDOM_MULTIPLIER 0x5DEECE66Dp-48
#define RANDOM_ADDEND 0xBp-48
#define RANDOM_SCALE 0x1p-48

inline uint random_next(Random *random, int bits) {
  *random = trunc((*random * RANDOM_MULTIPLIER + RANDOM_ADDEND) * RANDOM_SCALE);
  return (uint)((ulong)(*random / RANDOM_SCALE) >> (48 - bits));
}

#else

#define Random ulong
#define RANDOM_MULTIPLIER RANDOM_MULTIPLIER_LONG
#define RANDOM_ADDEND 0xBULL
#define RANDOM_MASK (1ULL << 48) - 1
#define RANDOM_SCALE 1

#define FAST_NEXT_INT

// Random::next(bits)
__host__ __device__ inline uint random_next(Random *random, int bits) {
    *random = (*random * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
    return (uint)(*random >> (48 - bits));
}
#endif // ~JRAND_DOUBLE

// new Random(seed)
#define get_random(seed) ((Random)((seed ^ RANDOM_MULTIPLIER_LONG) & RANDOM_MASK))
#define get_random_unseeded(state) ((Random) ((state) * RANDOM_SCALE))

// Random::nextInt(bound)
__host__ __device__ inline uint random_next_int(Random *random, uint bound) {
    int r = random_next(random, 31);
    int m = bound - 1;
    if ((bound & m) == 0) {
        r = (uint)((bound * (ulong)r) >> 31);
    } else {
#ifdef FAST_NEXT_INT
        r %= bound;
#else
        for (int u = r;
             u - (r = u % bound) + m < 0;
             u = random_next(random, 31));
#endif
    }
    return r;
}

__host__ __device__ inline int64_t random_next_long (Random *random) {
    return (((int64_t)random_next(random, 32)) << 32) + random_next(random, 32);
}

#define CHECK_GPU_ERR(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
        exit(code);
    }
}

// advance
#define advance(rand, multiplier, addend) ((rand) = ((rand) * (multiplier) + (addend)) & RANDOM_MASK)
#define advance_830(rand) advance(rand, 0x859D39E832D9LL, 0xE3E2DF5E9196LL)
#define advance_774(rand) advance(rand, 0xF8D900133F9LL, 0x5738CAC2F85ELL)
#define advance_387(rand) advance(rand, 0x5FE2BCEF32B5LL, 0xB072B3BF0CBDLL)
#define advance_16(rand) advance(rand, 0x6DC260740241LL, 0xD0352014D90LL)
#define advance_m1(rand) advance(rand, 0xDFE05BCB1365LL, 0x615C0E462AA9LL)
#define advance_m3759(rand) advance(rand, 0x63A9985BE4ADLL, 0xA9AA8DA9BC9BLL)



#define TREE_X 4
#define TREE_Z 3
#define TREE_HEIGHT 6

#define OTHER_TREE_COUNT 3
__device__ inline int getTreeHeight(int x, int z) {
    if (x == TREE_X && z == TREE_Z)
        return TREE_HEIGHT;

    if (x == 1 && z == 13)
        return 5;

    if (x == 6 && z == 12)
        return 6;

    if (x == 14 && z == 7) {
        return 5;
    }

    return 0;
}

#define WATERFALL_X 9
#define WATERFALL_Y 76
#define WATERFALL_Z 1



#define MODULUS (1LL << 48)
#define SQUARE_SIDE (MODULUS / 16)
#define X_TRANSLATE 0
#define Z_TRANSLATE 11
#define L00 7847617LL
#define L01 (-18218081LL)
#define L10 4824621LL
#define L11 24667315LL
#define LI00 (24667315.0 / 16)
#define LI01 (18218081.0 / 16)
#define LI10 (-4824621.0 / 16)
#define LI11 (7847617.0 / 16)

#define CONST_MIN(a, b) ((a) < (b) ? (a) : (b))
#define CONST_MIN4(a, b, c, d) CONST_MIN(CONST_MIN(a, b), CONST_MIN(c, d))
#define CONST_MAX(a, b) ((a) > (b) ? (a) : (b))
#define CONST_MAX4(a, b, c, d) CONST_MAX(CONST_MAX(a, b), CONST_MAX(c, d))
#define CONST_FLOOR(x) ((x) < (signed_seed_t) (x) ? (signed_seed_t) (x) - 1 : (signed_seed_t) (x))
#define CONST_CEIL(x) ((x) == (signed_seed_t) (x) ? (signed_seed_t) (x) : CONST_FLOOR((x) + 1))
#define CONST_LOWER(x, m, c) ((m) < 0 ? ((x) + 1 - (double) (c) / MODULUS) * (m) : ((x) - (double) (c) / MODULUS) * (m))
#define CONST_UPPER(x, m, c) ((m) < 0 ? ((x) - (double) (c) / MODULUS) * (m) : ((x) + 1 - (double) (c) / MODULUS) * (m))

// for a parallelogram ABCD https://media.discordapp.net/attachments/668607204009574411/671018577561649163/unknown.png
#define B_X LI00
#define B_Z LI10
#define C_X (LI00 + LI01)
#define C_Z (LI10 + LI11)
#define D_X LI01
#define D_Z LI11
#define LOWER_X CONST_MIN4(0, B_X, C_X, D_X)
#define LOWER_Z CONST_MIN4(0, B_Z, C_Z, D_Z)
#define UPPER_X CONST_MAX4(0, B_X, C_X, D_X)
#define UPPER_Z CONST_MAX4(0, B_Z, C_Z, D_Z)
#define ORIG_SIZE_X (UPPER_X - LOWER_X + 1)
#define SIZE_X CONST_CEIL(ORIG_SIZE_X - D_X)
#define SIZE_Z CONST_CEIL(UPPER_Z - LOWER_Z + 1)
#define TOTAL_WORK_SIZE (SIZE_X * SIZE_Z)

#define MAX_TREE_ATTEMPTS 12
#define MAX_TREE_SEARCH_BACK (3 * MAX_TREE_ATTEMPTS - 3 + 16 * OTHER_TREE_COUNT)

__constant__ ulong search_back_multipliers[MAX_TREE_SEARCH_BACK + 1];
__constant__ ulong search_back_addends[MAX_TREE_SEARCH_BACK + 1];
int search_back_count;

#define WORK_UNIT_SIZE (1LL << 23)
#define BLOCK_SIZE 256



__global__ void doWork(ulong offset, int* num_seeds, ulong* seeds, int gpu_search_back_count) {
    // lattice tree position
    ulong global_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset + global_id >= TOTAL_WORK_SIZE) return;

    signed_seed_t lattice_x = (signed_seed_t) ((offset + global_id) % SIZE_X) + LOWER_X;
    signed_seed_t lattice_z = (signed_seed_t) ((offset + global_id) / SIZE_X) + LOWER_Z;
    lattice_z += (B_X * lattice_z < B_Z * lattice_x) * SIZE_Z;
    if (D_X * lattice_z > D_Z * lattice_x) {
        lattice_x += B_X;
        lattice_z += B_Z;
    }
    lattice_x += (signed_seed_t) (TREE_X * LI00 + TREE_Z * LI01);
    lattice_z += (signed_seed_t) (TREE_X * LI10 + TREE_Z * LI11);
    Random rand = (Random) ((lattice_x * L00 + lattice_z * L01 + X_TRANSLATE) % MODULUS);
    advance_m1(rand);
    Random tree_start = rand;
    advance_m1(tree_start);

    bool res = random_next(&rand, 4) == TREE_X;
    res &= random_next(&rand, 4) == TREE_Z;
    res &= random_next_int(&rand, 3) == (ulong) (TREE_HEIGHT - 4);


    for (int treeBackCalls = 0; treeBackCalls <= gpu_search_back_count; treeBackCalls++) {
        Random start = (tree_start * search_back_multipliers[treeBackCalls] + search_back_addends[treeBackCalls]) & RANDOM_MASK;
        rand = start;

        bool this_res = res;
        this_res &= random_next_int(&rand, 10) != 0;

        bool generated_tree[16][16];
        memset(generated_tree, false, sizeof(generated_tree));

        int treesMatched = 0;
        bool any_population_matches = false;
        for (int treeAttempt = 0; treeAttempt <= MAX_TREE_ATTEMPTS; treeAttempt++) {
            int treeX = random_next(&rand, 4);
            int treeZ = random_next(&rand, 4);
            int wantedTreeHeight = getTreeHeight(treeX, treeZ);
            int treeHeight = random_next_int(&rand, 3) + 4;
            if (treeHeight == wantedTreeHeight && !generated_tree[treeX][treeZ]) {
                treesMatched++;
                generated_tree[treeX][treeZ] = true;
                advance_16(rand);
            }

            if (treesMatched == OTHER_TREE_COUNT + 1) {
                Random before_rest = rand;
                // yellow flowers
                advance_774(rand);
                // red flowers
                if (random_next(&rand, 1) == 0) {
                    advance_387(rand);
                }
                // brown mushroom
                if (random_next(&rand, 2) == 0) {
                    advance_387(rand);
                }
                // red mushroom
                if (random_next(&rand, 3) == 0) {
                    advance_387(rand);
                }
                // reeds
                advance_830(rand);
                // pumpkins
                if (random_next(&rand, 5) == 0) {
                    advance_387(rand);
                }

                for (int i = 0; i < 50; i++) {
                    bool waterfall_matches = random_next(&rand, 4) == WATERFALL_X;
                    waterfall_matches &= random_next_int(&rand, random_next_int(&rand, 120) + 8) == WATERFALL_Y;
                    waterfall_matches &= random_next(&rand, 4) == WATERFALL_Z;
                    any_population_matches |= waterfall_matches;
                }
                rand = before_rest;
            }
        }

        this_res &= any_population_matches;

        Random start_chunk_rand = start;
        advance_m3759(start_chunk_rand);
        if (this_res) {
            int index = atomicAdd(num_seeds, 1);
            seeds[index] = start_chunk_rand;
        }

        advance_m1(start);
    }
}

#define GPU_COUNT 1



struct GPU_Node {
    int GPU;
    int* num_seeds;
    ulong* seeds;
};
GPU_Node nodes[GPU_COUNT];

void setup_gpu_node(GPU_Node* node, int gpu) {
    CHECK_GPU_ERR(hipSetDevice(gpu));
    node->GPU = gpu;
    CHECK_GPU_ERR(hipMallocManaged(&node->num_seeds, sizeof(*node->num_seeds)));
    CHECK_GPU_ERR(hipMallocManaged(&node->seeds, (1LL << 10))); // approx 1kb
}


void calculate_search_backs() {
    bool allow_search_back[MAX_TREE_SEARCH_BACK + 1];
    memset(allow_search_back, false, sizeof(allow_search_back));

    for (int i = 0; i <= MAX_TREE_ATTEMPTS - OTHER_TREE_COUNT - 1; i++) {
        allow_search_back[i * 3] = true;
    }

    for (int tree = 0; tree < OTHER_TREE_COUNT; tree++) {
        for (int i = 0; i <= MAX_TREE_SEARCH_BACK - 19; i++) {
            if (allow_search_back[i])
                allow_search_back[i + 19] = true;
        }
    }

    search_back_count = 0;
    ulong multiplier = 1;
    ulong addend = 0;
    ulong multipliers[MAX_TREE_SEARCH_BACK + 1];
    ulong addends[MAX_TREE_SEARCH_BACK + 1];
    for (int i = 0; i <= MAX_TREE_SEARCH_BACK; i++) {
        if (allow_search_back[i]) {
            int index = search_back_count++;
            multipliers[index] = multiplier;
            addends[index] = addend;
        }
        multiplier = (multiplier * 0xDFE05BCB1365LL) & RANDOM_MASK;
        addend = (0xDFE05BCB1365LL * addend + 0x615C0E462AA9LL) & RANDOM_MASK;
    }

    for (int gpu = 0; gpu < GPU_COUNT; gpu++) {
        CHECK_GPU_ERR(hipSetDevice(gpu));
        CHECK_GPU_ERR(hipMemcpyToSymbol(HIP_SYMBOL(search_back_multipliers), &multipliers, search_back_count * sizeof(*multipliers)));
        CHECK_GPU_ERR(hipMemcpyToSymbol(HIP_SYMBOL(search_back_addends), &addends, search_back_count * sizeof(*addends)));
    }
}


#undef int
int main() {
#define int int32_t
    printf("Searching %lld total seeds...\n", TOTAL_WORK_SIZE);

    calculate_search_backs();

    FILE* out_file = fopen("chunk_seeds.txt", "w");

    for(int i = 0; i < GPU_COUNT; i++) {
        setup_gpu_node(&nodes[i],i);
    }

    
    ulong count = 0;
    clock_t lastIteration = clock();
    clock_t startTime = clock();
    for (ulong offset = 0; offset < TOTAL_WORK_SIZE;) {
        
        for(int gpu_index = 0; gpu_index < GPU_COUNT; gpu_index++) {
            CHECK_GPU_ERR(hipSetDevice(gpu_index));
            *nodes[gpu_index].num_seeds = 0;
            doWork <<<WORK_UNIT_SIZE / BLOCK_SIZE, BLOCK_SIZE>>> (offset, nodes[gpu_index].num_seeds, nodes[gpu_index].seeds, search_back_count);
            offset += WORK_UNIT_SIZE;
        }
        
        for(int gpu_index = 0; gpu_index < GPU_COUNT; gpu_index++) {
            CHECK_GPU_ERR(hipSetDevice(gpu_index));
            CHECK_GPU_ERR(hipDeviceSynchronize());
            
            for (int i = 0, e = *nodes[gpu_index].num_seeds; i < e; i++) {
                fprintf(out_file, "%lld\n", nodes[gpu_index].seeds[i]);
            }
            fflush(out_file);
            count += *nodes[gpu_index].num_seeds;
        }
        
        double iterationTime = (double)(clock() - lastIteration) / CLOCKS_PER_SEC;
        double timeElapsed = (double)(clock() - startTime) / CLOCKS_PER_SEC;
        lastIteration = clock();
        ulong numSearched = offset + WORK_UNIT_SIZE;
        double speed = (double)WORK_UNIT_SIZE / (double)iterationTime / 1000000.0;
        double progress = (double)numSearched / (double)TOTAL_WORK_SIZE * 100.0;
        double estimatedTime = (double)(TOTAL_WORK_SIZE - numSearched) / (double) WORK_UNIT_SIZE * iterationTime / 3600.0;
        printf("Searched: %lld seeds. Found: %lld matches. Uptime: %.1fs. Speed: %.2fm seeds/s. Completion: %.3f%%. ETA: %.1fh.\n", numSearched, count, timeElapsed, speed, progress, estimatedTime);
    }

    fclose(out_file);

}
