
// IDE indexing
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __shared__
#define __constant__
#define __global__
#define __CUDACC__
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_cmath.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <device_functions.h>
#endif


#include <hip/hip_runtime.h>
#include <ctype.h>
#include <memory.h>
#include <stdint.h>
#include <stdio.h>
#include <time.h>

#define signed_seed_t int64_t
#define uint uint32_t
#define ulong uint64_t
// let's be EVIL (and make sure all includes come before this)
#define int int32_t

/* JAVA RANDOM EMULATION */
#undef JRAND_DOUBLE

#define RANDOM_MULTIPLIER_LONG 0x5DEECE66DULL

#ifdef JRAND_DOUBLE
#define Random double
#define RANDOM_MULTIPLIER 0x5DEECE66Dp-48
#define RANDOM_ADDEND 0xBp-48
#define RANDOM_SCALE 0x1p-48

inline uint __host__ __device__ random_next(Random* random, int bits) {
    *random = trunc((*random * RANDOM_MULTIPLIER + RANDOM_ADDEND) * RANDOM_SCALE);
    return (uint)((ulong)(*random / RANDOM_SCALE) >> (48 - bits));
}

#else

#define Random ulong
#define RANDOM_MULTIPLIER RANDOM_MULTIPLIER_LONG
#define RANDOM_ADDEND 0xBULL
#define RANDOM_MASK (1ULL << 48) - 1
#define RANDOM_SCALE 1

#define FAST_NEXT_INT

// Random::next(bits)
__host__ __device__ inline uint random_next(Random *random, int bits) {
    *random = (*random * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
    return (uint)(*random >> (48 - bits));
}
#endif // ~JRAND_DOUBLE

// new Random(seed)
#define get_random(seed) ((Random)((seed ^ RANDOM_MULTIPLIER_LONG) & RANDOM_MASK))
#define get_random_unseeded(state) ((Random)((state)*RANDOM_SCALE))

// Random::nextInt(bound)
__host__ __device__ inline uint random_next_int(Random *random, uint bound) {
    int r = random_next(random, 31);
    int m = bound - 1;
    if ((bound & m) == 0) {
        // Could probably use __mul64hi here
        r = (uint)((bound * (ulong)r) >> 31);
    } else {
#ifdef FAST_NEXT_INT
        r %= bound;
#else
        for (int u = r;
             u - (r = u % bound) + m < 0;
             u = random_next(random, 31));
#endif
    }
    return r;
}

__host__ __device__ inline int64_t random_next_long(Random *random) {
    return (((int64_t)random_next(random, 32)) << 32) + random_next(random, 32);
}

/* Random advance */
#define advance(rand, multiplier, addend) ((rand) = ((rand) * (multiplier) + (addend)) & RANDOM_MASK)
#define advance_830(rand) advance(rand, 0x859D39E832D9LL, 0xE3E2DF5E9196LL)
#define advance_774(rand) advance(rand, 0xF8D900133F9LL, 0x5738CAC2F85ELL)
#define advance_387(rand) advance(rand, 0x5FE2BCEF32B5LL, 0xB072B3BF0CBDLL)
#define advance_16(rand) advance(rand, 0x6DC260740241LL, 0xD0352014D90LL)
#define advance_m1(rand) advance(rand, 0xDFE05BCB1365LL, 0x615C0E462AA9LL)
#define advance_m3760(rand) advance(rand, 0x2DBFC2690F41LL, 0xB26D332614D0LL)

/* LATTICE CONSTANTS */
#define MODULUS (1LL << 48)
#define SQUARE_SIDE (MODULUS / 16)
#define X_TRANSLATE 0
#define Z_TRANSLATE 11
#define L00 7847617LL
#define L01 (-18218081LL)
#define L10 4824621LL
#define L11 24667315LL
#define LI00 (24667315.0 / 16)
#define LI01 (18218081.0 / 16)
#define LI10 (-4824621.0 / 16)
#define LI11 (7847617.0 / 16)

#define CONST_MIN(a, b) ((a) < (b) ? (a) : (b))
#define CONST_MIN4(a, b, c, d) CONST_MIN(CONST_MIN(a, b), CONST_MIN(c, d))
#define CONST_MAX(a, b) ((a) > (b) ? (a) : (b))
#define CONST_MAX4(a, b, c, d) CONST_MAX(CONST_MAX(a, b), CONST_MAX(c, d))
#define CONST_FLOOR(x) ((x) < (signed_seed_t)(x) ? (signed_seed_t)(x)-1 : (signed_seed_t)(x))
#define CONST_CEIL(x) ((x) == (signed_seed_t)(x) ? (signed_seed_t)(x) : CONST_FLOOR((x) + 1))
#define CONST_LOWER(x, m, c) ((m) < 0 ? ((x) + 1 - (double)(c) / MODULUS) * (m) : ((x) - (double)(c) / MODULUS) * (m))
#define CONST_UPPER(x, m, c) ((m) < 0 ? ((x) - (double)(c) / MODULUS) * (m) : ((x) + 1 - (double)(c) / MODULUS) * (m))

// for a parallelogram ABCD https://media.discordapp.net/attachments/668607204009574411/671018577561649163/unknown.png
#define B_X LI00
#define B_Z LI10
#define C_X (LI00 + LI01)
#define C_Z (LI10 + LI11)
#define D_X LI01
#define D_Z LI11
#define LOWER_X CONST_MIN4(0, B_X, C_X, D_X)
#define LOWER_Z CONST_MIN4(0, B_Z, C_Z, D_Z)
#define UPPER_X CONST_MAX4(0, B_X, C_X, D_X)
#define UPPER_Z CONST_MAX4(0, B_Z, C_Z, D_Z)
#define ORIG_SIZE_X (UPPER_X - LOWER_X + 1)
#define SIZE_X CONST_CEIL(ORIG_SIZE_X - D_X)
#define SIZE_Z CONST_CEIL(UPPER_Z - LOWER_Z + 1)
#define TOTAL_WORK_SIZE (SIZE_X * SIZE_Z)

#define WORK_UNIT_SIZE (1LL << 23)
#define BLOCK_SIZE 256

/* TREE/WATERFALL CONSTANTS */
#define TREE_X 4
#define TREE_Z 3
#define TREE_HEIGHT 6
#define OTHER_TREE_COUNT 3

#define WATERFALL_X 9
#define WATERFALL_Y 76
#define WATERFALL_Z 1

#define MAX_TREE_ATTEMPTS 12
#define MAX_TREE_SEARCH_BACK (3 * MAX_TREE_ATTEMPTS - 3 + 16 * OTHER_TREE_COUNT)

__device__ inline int getTreeHeight(int x, int z) {
    if (x == TREE_X && z == TREE_Z)
        return TREE_HEIGHT;

    if (x == 1 && z == 13)
        return 5;

    if (x == 6 && z == 12)
        return 6;

    if (x == 14 && z == 7) {
        return 5;
    }

    return 0;
}

__constant__ ulong search_back_multipliers[MAX_TREE_SEARCH_BACK + 1];
__constant__ ulong search_back_addends[MAX_TREE_SEARCH_BACK + 1];
int search_back_count;

// Adds to tree flags any tree matching the parameters, returns whether a tree was spawned
__device__ inline bool addTreeFlags(int* flags, int x, int z, int height) {
    int old_flags = *flags;
    *flags |= (x == TREE_X && z == TREE_Z && height == TREE_HEIGHT);
    *flags |= (x == WATERFALL_X - 3 && z == WATERFALL_Z + 3 && height == 5) << 1;
    *flags |= ((x >= WATERFALL_X + 3 && x <= WATERFALL_X + 5) && (z <= WATERFALL_Z - 6 && z >= WATERFALL_Z - 9) && (height == 4 || height == 5)) << 2;
    return *flags != old_flags;
}

__global__ void doCalcTreeStarts(ulong offset,
                                 Random* tree_starts,
                                 int* num_tree_starts) {
    // lattice tree position
    ulong global_id = blockIdx.x * blockDim.x + threadIdx.x;

    signed_seed_t lattice_x = (signed_seed_t)((offset + global_id) % SIZE_X) + LOWER_X;
    signed_seed_t lattice_z = (signed_seed_t)((offset + global_id) / SIZE_X) + LOWER_Z;
    lattice_z += (B_X * lattice_z < B_Z * lattice_x) * SIZE_Z;
    if (D_X * lattice_z > D_Z * lattice_x) {
        lattice_x += B_X;
        lattice_z += B_Z;
    }
    lattice_x += (signed_seed_t)(TREE_X * LI00 + TREE_Z * LI01);
    lattice_z += (signed_seed_t)(TREE_X * LI10 + TREE_Z * LI11);

    Random rand = (Random)((lattice_x * L00 + lattice_z * L01 + X_TRANSLATE) % MODULUS);
    advance_m1(rand);

    Random tree_start = rand;
    advance_m1(tree_start);

    bool res = random_next(&rand, 4) == TREE_X;
    res &= random_next(&rand, 4) == TREE_Z;
    res &= random_next_int(&rand, 3) == (ulong)(TREE_HEIGHT - 4);

    if (res) {
        int index = atomicAdd(num_tree_starts, 1);
        tree_starts[index] = tree_start;
    }
}

__global__ void doCalcRandStarts(int* num_tree_starts,
                                 Random* tree_starts,
                                 int* num_rand_starts,
                                 Random* rand_starts,
                                 int gpu_search_back_count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < *num_tree_starts; i += blockDim.x * gridDim.x) {
        Random tree_start = tree_starts[i];

        for (int treeBackCalls = 0; treeBackCalls <= gpu_search_back_count; treeBackCalls++) {
            ulong multiplier = search_back_multipliers[treeBackCalls];
            ulong addend = search_back_addends[treeBackCalls];

            Random rand = (tree_start * multiplier + addend) & RANDOM_MASK;
            if (random_next_int(&rand, 10) != 0) {
                int index = atomicAdd(num_rand_starts, 1);
                rand_starts[index] = rand;
            }
        }
    }
}

__global__ void doCalcChunkSeed(int* num_rand_starts, 
                                Random* rand_starts,
                                int* num_seeds,
                                ulong* seeds) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < *num_rand_starts; i += blockDim.x * gridDim.x) {
        Random rand = rand_starts[i];

        char generated_tree[16][2];
        memset(generated_tree, 0x00, sizeof(generated_tree));

        int treesMatched = 0;
        bool any_population_matches = false;
        for (int treeAttempt = 0; treeAttempt <= MAX_TREE_ATTEMPTS; treeAttempt++) {
            int treeX = random_next(&rand, 4);
            int treeZ = random_next(&rand, 4);
            int wantedTreeHeight = getTreeHeight(treeX, treeZ);
            int treeHeight = random_next_int(&rand, 3) + 4;

            char& boolpack = generated_tree[treeX][treeZ / 2];
            const char mask = 1 << (treeZ % 8);

            if (treeHeight == wantedTreeHeight && !(boolpack & mask)) {
                treesMatched++;
                boolpack |= mask;
                advance_16(rand);
            }

			if (treesMatched == OTHER_TREE_COUNT + 1) {
                Random before_rest = rand;
                // yellow flowers
                advance_774(rand);
                // red flowers
                if (random_next(&rand, 1) == 0) advance_387(rand);
                // brown mushroom
                if (random_next(&rand, 2) == 0) advance_387(rand);
                // red mushroom
                if (random_next(&rand, 3) == 0) advance_387(rand);

                // reeds
                advance_830(rand);
                // pumpkins
                if (random_next(&rand, 5) == 0) advance_387(rand);

                for (int i = 0; i < 50; i++) {
                    bool waterfall_matches = random_next(&rand, 4) == WATERFALL_X;
                    waterfall_matches &= random_next_int(&rand, random_next_int(&rand, 120) + 8) == WATERFALL_Y;
                    waterfall_matches &= random_next(&rand, 4) == WATERFALL_Z;
                    any_population_matches |= waterfall_matches;
                }
                rand = before_rest;
            }
        }

        if (any_population_matches) {
            Random start_chunk_rand = rand;
            advance_m3760(start_chunk_rand);

            int index = atomicAdd(num_seeds, 1);
            seeds[index] = start_chunk_rand;
        }
    }
}

#define CHECK_GPU_ERR(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
        exit(code);
    }
}

struct GPU_Node {
    int GPU;

    int* num_seeds;
    ulong* seeds;

    int* num_tree_starts;
    Random* tree_starts;

    int* num_rand_starts;
    Random* rand_starts;
};

void setup_gpu_node(GPU_Node* node, int gpu) {
    CHECK_GPU_ERR(hipSetDevice(gpu));
    node->GPU = gpu;
    CHECK_GPU_ERR(hipMallocManaged(&node->num_seeds, sizeof(*node->num_seeds)));
    CHECK_GPU_ERR(hipMallocManaged(&node->seeds, (1LL << 20))); // approx 1MB

    CHECK_GPU_ERR(hipMallocManaged(&node->num_tree_starts, sizeof(*node->num_tree_starts)));
    CHECK_GPU_ERR(hipMallocManaged(&node->tree_starts, (sizeof(Random)*WORK_UNIT_SIZE)));

    CHECK_GPU_ERR(hipMallocManaged(&node->num_rand_starts, sizeof(*node->num_rand_starts)));
    CHECK_GPU_ERR(hipMallocManaged(&node->rand_starts, (sizeof(Random)*WORK_UNIT_SIZE*10000)));
}

void calculate_search_backs(int GPU_COUNT) {
    bool allow_search_back[MAX_TREE_SEARCH_BACK + 1];
    memset(allow_search_back, false, sizeof(allow_search_back));

    for (int i = 0; i <= MAX_TREE_ATTEMPTS - OTHER_TREE_COUNT - 1; i++) {
        allow_search_back[i * 3] = true;
    }

    for (int tree = 0; tree < OTHER_TREE_COUNT; tree++) {
        for (int i = 0; i <= MAX_TREE_SEARCH_BACK - 19; i++) {
            if (allow_search_back[i])
                allow_search_back[i + 19] = true;
        }
    }

    search_back_count = 0;
    ulong multiplier = 1;
    ulong addend = 0;
    ulong multipliers[MAX_TREE_SEARCH_BACK + 1];
    ulong addends[MAX_TREE_SEARCH_BACK + 1];
    for (int i = 0; i <= MAX_TREE_SEARCH_BACK; i++) {
        if (allow_search_back[i]) {
            int index = search_back_count++;
            multipliers[index] = multiplier;
            addends[index] = addend;
        }
        multiplier = (multiplier * 0xDFE05BCB1365LL) & RANDOM_MASK;
        addend = (0xDFE05BCB1365LL * addend + 0x615C0E462AA9LL) & RANDOM_MASK;
    }

    for (int gpu = 0; gpu < GPU_COUNT; gpu++) {
        CHECK_GPU_ERR(hipSetDevice(gpu));
        CHECK_GPU_ERR(hipMemcpyToSymbol(HIP_SYMBOL(search_back_multipliers), &multipliers, search_back_count * sizeof(*multipliers)));
        CHECK_GPU_ERR(hipMemcpyToSymbol(HIP_SYMBOL(search_back_addends), &addends, search_back_count * sizeof(*addends)));
    }
}

#undef int
int main(int argc, char *argv[]) {
#define int int32_t
    int GPU_COUNT = 1;
    for (int i = 1; i < argc; i++) {
        if (argv[i][0] == '-') {
            switch (argv[i][1]) {
                case 'g':
                    if (isdigit(argv[i][2])) GPU_COUNT = atoi(argv[i] + 2);
                    break;
                default:
                    printf("Error: Flag not recognized.");
                    return -1;
            }
        } else {
            printf("Error: Please specify flag before argument.");
            return -1;
        }
    }
    GPU_Node *nodes = (GPU_Node*)malloc(sizeof(GPU_Node) * GPU_COUNT);
    printf("Searching %lld total seeds...\n", TOTAL_WORK_SIZE);

    calculate_search_backs(GPU_COUNT);

    FILE* out_file = fopen("chunk_seeds.txt", "w");

    for (int i = 0; i < GPU_COUNT; i++) {
        setup_gpu_node(&nodes[i], i);
    }

    ulong count = 0;
    clock_t lastIteration = clock();
    clock_t startTime = clock();
    for (ulong offset = 0; offset < TOTAL_WORK_SIZE;) {
        for (int gpu_index = 0; gpu_index < GPU_COUNT; gpu_index++) {
            CHECK_GPU_ERR(hipSetDevice(gpu_index));

            GPU_Node& node = nodes[gpu_index];

            *node.num_tree_starts = 0;
            doCalcTreeStarts <<<WORK_UNIT_SIZE / BLOCK_SIZE, BLOCK_SIZE>>> (offset,
                    node.tree_starts,
                    node.num_tree_starts);
            offset += WORK_UNIT_SIZE;

            *node.num_rand_starts = 0;
            doCalcRandStarts <<<WORK_UNIT_SIZE / BLOCK_SIZE, BLOCK_SIZE>>> (node.num_tree_starts,
                    node.tree_starts,
                    node.num_rand_starts,
                    node.rand_starts,
                    search_back_count);

            *node.num_seeds = 0;
            doCalcChunkSeed <<<WORK_UNIT_SIZE / BLOCK_SIZE, BLOCK_SIZE>>> (node.num_rand_starts, 
                    node.rand_starts, 
                    node.num_seeds,
                    node.seeds);
        }

        for (int gpu_index = 0; gpu_index < GPU_COUNT; gpu_index++) {
            CHECK_GPU_ERR(hipSetDevice(gpu_index));
            CHECK_GPU_ERR(hipDeviceSynchronize());

            for (int i = 0, e = *nodes[gpu_index].num_seeds; i < e; i++) {
                fprintf(out_file, "%lld\n", nodes[gpu_index].seeds[i]);
            }
            fflush(out_file);
            count += *nodes[gpu_index].num_seeds;
        }

        double iterationTime = (double)(clock() - lastIteration) / CLOCKS_PER_SEC;
        double timeElapsed = (double)(clock() - startTime) / CLOCKS_PER_SEC;
        lastIteration = clock();
        ulong numSearched = offset + WORK_UNIT_SIZE * GPU_COUNT;
        double speed = (double)WORK_UNIT_SIZE * GPU_COUNT / (double)iterationTime / 1000000.0;
        double progress = (double)numSearched / (double)TOTAL_WORK_SIZE * 100.0;
        double estimatedTime = (double)(TOTAL_WORK_SIZE - numSearched) / (double)(WORK_UNIT_SIZE * GPU_COUNT) * iterationTime;
        char suffix = 's';
        if (estimatedTime >= 3600) {
            suffix = 'h';
            estimatedTime /= 3600.0;
        } else if (estimatedTime >= 60) {
            suffix = 'm';
            estimatedTime /= 60.0;
        }
        if (progress >= 100.0) {
            estimatedTime = 0.0;
            suffix = 's';
        }
        printf("Searched: %lld seeds. Found: %lld matches. Uptime: %.1fs. Speed: %.2fm seeds/s. Completion: %.3f%%. ETA: %.1f%c.\n", numSearched, count, timeElapsed, speed, progress, estimatedTime, suffix);
    }
    fclose(out_file);
}
